
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

# define THREADS_PER_BLOCK 1024

double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}


// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, const char *filename, int *num_rows, int *num_cols, int *num_vals) {
    
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    
    // Get number of rows, columns, and non-zero values
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int *row_ptr_t = (int *) malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *) malloc(*num_vals * sizeof(int));
    float *values_t = (float *) malloc(*num_vals * sizeof(float));
    
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *) malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++) {
        row_occurances[i] = 0;
    }
    
    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        
        row_occurances[row]++;
    }
    
    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++) {
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);
    
    // Set the file position to the beginning of the file
    rewind(file);
    
    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++) {
        col_ind_t[i] = -1;
    }
    
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int i = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        row--;
        column--;
        
        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1) {
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        i = 0;
    }
    
    fclose(file);
    
    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;
}

// CPU implementation of SPMV using CSR
void spmv_csr_sw(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, const float *x, float *y) {
    for (int i = 0; i < num_rows; i++) {
        float dotProduct = 0;
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        
        for (int j = row_start; j < row_end; j++) {
            dotProduct += values[j] * x[col_ind[j]];
        }
        
        y[i] = dotProduct;
    }
}

// function to check the CUDAMalloc allocation of mamory in GPU
#define CHECK(call){                                                                    \
    const hipError_t err = call;                                                       \
    if (err != hipSuccess) {                                                           \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}                                                                                       \

// function to ckeck the call to the Kernel CUDA to be executed on GPU
#define CHECK_KERNELCALL(){                                                             \
    const hipError_t err = hipGetLastError();                                         \
    if (err != hipSuccess) {                                                           \
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);   \
        exit(EXIT_FAILURE);                                                             \
    }                                                                                   \
}

// kernel to multiply the values in the matrix and in the vector
__global__ void spmv_csr_mul(const int *col_ind, float *values, const float *vec, int num_vals){
    int i = blockIdx.x * blockDim.x + threadIdx.x;  
    if(i<num_vals)
        values[i] = vec[col_ind[i]] * values[i];
}

// kernel to inizilise a vector
__global__ void vector_inizialiser(float *vector, int dim){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i<dim)
        vector[i] = 0;
}

// kernel to sum the rows of the matrix
__global__ void parallel_sum(float *input, float *output, int dim_slice, int out_index, int start_slice){    
    __shared__ float shared_input[THREADS_PER_BLOCK * sizeof(float)];
    int t_id = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < dim_slice){
        shared_input[t_id] = input[index + start_slice];
        __syncthreads();

        for (unsigned int j=1; j < dim_slice; j *= 2) {
            int i = 2 * j * t_id;
            if (i < blockDim.x) {
                shared_input[i] += shared_input[i + j];
            }
            __syncthreads();
            }

        if (t_id == 0){
            output[out_index] += shared_input[0];
        }
    }
}

int main(int argc, const char * argv[]) {

    if (argc != 2) {
        printf("Usage: ./exec matrix_file\n");
        return 0;
    }
    
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals;
    float *values;
    
    const char *filename = argv[1];

    double start_cpu, end_cpu;
    
    read_matrix(&row_ptr, &col_ind, &values, filename, &num_rows, &num_cols, &num_vals);
    
    float *x = (float *) malloc(num_rows * sizeof(float));
    float *y_sw = (float *) malloc(num_rows * sizeof(float));

    // Generate a random vector

    srand(time(NULL));

    for (int i = 0; i < num_rows; i++) {
        x[i] = (rand()%100)/(rand()%100+1); //the number we use to divide cannot be 0, that's the reason of the +1 
    }

    // Compute in sw
    start_cpu = get_time();
    spmv_csr_sw(row_ptr, col_ind, values, num_rows, x, y_sw);
    end_cpu = get_time();

    // Decleare GPU var for CSR

    int *col_gpu;
    float *values_gpu;
    double start_gpu, end_gpu, gpu_time;

    CHECK(hipMalloc((int**)&col_gpu, num_vals * sizeof(float)));
    CHECK(hipMalloc((float**)&values_gpu, num_vals * sizeof(float)));

    CHECK(hipMemcpy(col_gpu, col_ind, num_vals * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(values_gpu, values, num_vals * sizeof(float), hipMemcpyHostToDevice));
    
    // Declear GPU var for input and output vector
    float *x_gpu, *y_gpu;

    CHECK(hipMalloc((float**)&x_gpu, num_rows * sizeof(float)));
    CHECK(hipMalloc((float**)&y_gpu, num_rows * sizeof(float)));
    
    CHECK(hipMemcpy(x_gpu, x, num_rows * sizeof(float), hipMemcpyHostToDevice));

    
    /* ----- Execution on GPU ----- */

    dim3 blockPerGrid((num_vals/THREADS_PER_BLOCK) + 1,1,1);
    dim3 threadsPerBlock(THREADS_PER_BLOCK,1,1);

    
    // FIRST Kernel for moltiplications
    start_gpu = get_time();
    spmv_csr_mul <<<blockPerGrid,threadsPerBlock>>>(col_gpu, values_gpu, x_gpu, num_vals);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    end_gpu = get_time();
    gpu_time = end_gpu - start_gpu;


    // SECOND Kernel for sum parallel reduction

    // inizialise the result vector
    blockPerGrid.x = (num_rows * sizeof(float) / THREADS_PER_BLOCK) + 1;
    start_gpu = get_time();
    vector_inizialiser <<<blockPerGrid,threadsPerBlock>>>(y_gpu, num_rows * sizeof(float));
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    end_gpu = get_time();
    gpu_time += end_gpu - start_gpu;

    // parallel sum
    int slice_start, slice_end, slice_dim;

    for(int i = 0; i < num_rows + 1; i++){
        slice_start = row_ptr[i];
        slice_end = row_ptr[i+1];
        slice_dim = slice_end - slice_start;
        blockPerGrid.x = ((slice_dim/THREADS_PER_BLOCK) + 1);
        start_gpu = get_time();
        parallel_sum<<<blockPerGrid,threadsPerBlock>>>(values_gpu, y_gpu, slice_dim, i, slice_start);
        CHECK_KERNELCALL();
        CHECK(hipDeviceSynchronize());
        end_gpu = get_time();
        gpu_time += end_gpu - start_gpu;
    }
    
    float *y_hw = (float *) malloc(num_rows * sizeof(float));
    CHECK(hipMemcpy(y_hw, y_gpu, num_rows * sizeof(float), hipMemcpyDeviceToHost));

    // check differences
    float diff = 0;

    for(int i = 0; i < num_rows; i++)
        diff = y_sw[i] - y_hw[i];   
    
    printf("tot difference: %d\n", diff);

    // Print time
    printf("SPMV Time CPU: %.10lf\n", end_cpu - start_cpu);
    printf("SPMV Time GPU: %.10lf\n", gpu_time);

    // Free    
    free(row_ptr);
    free(col_ind);
    free(values);
    free(y_sw);

    CHECK(hipFree(col_gpu));
    CHECK(hipFree(values_gpu));
    CHECK(hipFree(x_gpu));
    CHECK(hipFree(y_gpu));

    return 0;
}
